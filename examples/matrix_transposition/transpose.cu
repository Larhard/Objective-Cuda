#include "hip/hip_runtime.h"
#include "transpose.h"

extern "C" {
    __global__
        void transpose_naive(int *dest, const int *src, const int height, const int width) {
            int x = (blockIdx.x * blockDim.x) + threadIdx.x;
            int y = (blockIdx.y * blockDim.y) + threadIdx.y;

            dest[x*height+y] = src[y*width+x];
        }

    __global__
        void transpose_with_shared_mem(int* dest, const int* src) {
            __shared__ int tmp[TILE_DIM][TILE_DIM+1];
            /* int x = (blockIdx.x * blockDim.x); */
            /* int y = (blockIdx.y * blockDim.x); */

            int width = gridDim.x * TILE_DIM;
            int height = gridDim.y * TILE_DIM;

            int x = (blockIdx.x * blockDim.x) + threadIdx.x;
            int y = (blockIdx.y * blockDim.x);

            for (int i = threadIdx.y; i < blockDim.x; i += blockDim.y) {
                tmp[i][threadIdx.x] = src[(y + i) * width + x];
            }
            __syncthreads();

            x = (blockIdx.x * blockDim.x);
            y = (blockIdx.y * blockDim.x) + threadIdx.x;

            for (int i = threadIdx.y; i < blockDim.x; i += blockDim.y) {
                dest[(x + i) * height + y] = tmp[threadIdx.x][i];
            }
        }
}
